#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>


#define N 10

void add( int *a, int *b, int *c ) {
    int tid = 0; // this is CPU zero, so we start at zero
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += 1; // we have one CPU, so we increment by one
    }
}

__global__ void add_gpu( int *a, int *b, int *c ) {
    int tid = blockIdx.x; // handle the data at this index
    if (tid < N)
        c[tid] = a[tid] + b[tid];
}

#define HANDLE_ERROR(e)  _HANDLE_ERROR(e, __LINE__)

void _HANDLE_ERROR(hipError_t e, int line)
{
    if (e != hipSuccess)
    {
        printf("line: %d. error %s\n", line, hipGetErrorString(e));
        exit (1);
    }
}

int main()
{
    int a[N], b[N], c[N];
    int * dev_a, * dev_b, * dev_c;
    int i;

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    for (i=0; i<N; i++) {
        a[i] = -i;
        b[i] = i * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                hipMemcpyHostToDevice ) );

    //add(a,b,c);
    add_gpu<<<N,1>>>( dev_a, dev_b, dev_c );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
           hipMemcpyDeviceToHost) );

    // display the results
    for (i=0; i<N; i++) {
        printf( "%d + %d = %d\n", a[i], b[i], c[i] );
    }
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    
    return 0;
}
